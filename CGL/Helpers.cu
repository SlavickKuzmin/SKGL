#include "hip/hip_runtime.h"
#include "Helpers.cuh"

void cudasafe(int error, char* message, char* file, int line) 
{
	if (error != hipSuccess) {
		fprintf(stderr, "CUDA Error: %s : %i. In %s line %d\n", message, error, file, line);
		exit(-1);
	}
}

void printDeviceInfo()
{
	int deviceCount;

	cudasafe(hipGetDeviceCount(&deviceCount), "GetDeviceCount", (char*)__FILE__, __LINE__);

	printf("Number of CUDA devices %d.\n", deviceCount);

	for (int dev = 0; dev < deviceCount; dev++) {
		hipDeviceProp_t deviceProp;

		cudasafe(hipGetDeviceProperties(&deviceProp, dev), "Get Device Properties", (char*)__FILE__, __LINE__);

		if (dev == 0) {
			if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
				printf("No CUDA GPU has been detected\n");
				return;
			}
			else if (deviceCount == 1) {
				printf("There is 1 device supporting CUDA\n");
			}
			else {
				printf("There are %d devices supporting CUDA\n", deviceCount);
			}
		}

		printf("For device #%d\n", dev);
		printf("Device name:                %s\n", deviceProp.name);
		printf("Major revision number:      %d\n", deviceProp.major);
		printf("Minor revision Number:      %d\n", deviceProp.minor);
		printf("Total Global Memory:        %u\n", deviceProp.totalGlobalMem);
		printf("Total shared mem per block: %d\n", deviceProp.sharedMemPerBlock);
		printf("Total const mem size:       %d\n", deviceProp.totalConstMem);
		printf("Warp size:                  %d\n", deviceProp.warpSize);
		printf("Maximum block dimensions:   %d x %d x %d\n", deviceProp.maxThreadsDim[0], \
			deviceProp.maxThreadsDim[1], \
			deviceProp.maxThreadsDim[2]);

		printf("Maximum grid dimensions:    %d x %d x %d\n", deviceProp.maxGridSize[0], \
			deviceProp.maxGridSize[1], \
			deviceProp.maxGridSize[2]);
		printf("Clock Rate:                 %d\n", deviceProp.clockRate);
		printf("Number of muliprocessors:   %zd\n", deviceProp.multiProcessorCount);
		printf("\nPress any key to continue...\n");
		getchar();
	}
}

__device__ void setPixel(void* pixels, int pinch, int x, int y, Color color)
{
	Uint8 *pixel = (Uint8*)pixels;
	pixel += ((800-y) * pinch) + (x * sizeof(Uint32));
	*((Uint32*)pixel) = packColorToUint32(color);//abgr
}

__device__ void swap(int &x, int &y)
{
	int tmp = x;
	x = y;
	y = tmp;
}

__device__ void swapVec2i(Vec2i &x, Vec2i &y)
{
	Vec2i tmp = x;
	x = y;
	y = tmp;
}

__device__ void triangle(Vec2i t0, Vec2i t1, Vec2i t2, void* pixels, int pinch, Color *col) {
	if (t0.y == t1.y && t0.y == t2.y) return; // i dont care about degenerate triangles
	// sort the vertices, t0, t1, t2 lower-to-upper (bubblesort yay!)
	if (t0.y > t1.y) swapVec2i(t0, t1);
	if (t0.y > t2.y) swapVec2i(t0, t2);
	if (t1.y > t2.y) swapVec2i(t1, t2);
	int total_height = t2.y - t0.y;
	for (int i = 0; i < total_height; i++) {
		bool second_half = i > t1.y - t0.y || t1.y == t0.y;
		int segment_height = second_half ? t2.y - t1.y : t1.y - t0.y;
		float alpha = (float)i / total_height;
		float beta = (float)(i - (second_half ? t1.y - t0.y : 0)) / segment_height; // be careful: with above conditions no division by zero here
		Vec2i A = t0 + (t2 - t0)*alpha;
		Vec2i B = second_half ? t1 + (t2 - t1)*beta : t0 + (t1 - t0)*beta;
		if (A.x > B.x) swapVec2i(A, B);
		for (int j = A.x; j <= B.x; j++) {
			//image.set(j, t0.y + i, color); // attention, due to int casts t0.y+i != A.y
			setPixel(pixels, pinch, j, t0.y + i, *col);
		}
	}
}

__device__ void line(int x0, int y0, int x1, int y1, void* pixels, int pinch, Color *col) {
	bool steep = false;
	if (abs(x0 - x1) < abs(y0 - y1)) { // if the line is steep, we transpose the image
		swap(x0, y0);
		swap(x1, y1);
		steep = true;
	}
	if (x0 > x1) { // make it left-to-right
		swap(x0, x1);
		swap(y0, y1);
	}
	for (int x = x0; x <= x1; x++) {
		float t = (x - x0) / (float)(x1 - x0);
		int y = y0 * (1. - t) + y1 * t;
		if (steep) {
			setPixel(pixels, pinch, y, x, *col); // if transposed, de-transpose
		}
		else {
			setPixel(pixels, pinch, x, y, *col);
		}
	}
}