#include "hip/hip_runtime.h"
#include "Shader.cuh"

__device__ Vec4f Shader::vertex(int iface, int nthvert, Matrix ModelView, Matrix Projection, Vec2f uv, Vec3f normal, Vec3f vert) {
	varying_uv.set_col(nthvert, uv);// model->uv(iface, nthvert));
	varying_nrm.set_col(nthvert, proj<3>((Projection*ModelView).invert_transpose()*embed<4>(normal, 0.f)));//model->normal(iface, nthvert)
	Vec4f gl_Vertex = Projection * ModelView*embed<4>(vert);//model->vert(iface, nthvert));
	varying_tri.set_col(nthvert, gl_Vertex);
	ndc_tri.set_col(nthvert, proj<3>(gl_Vertex / gl_Vertex[3]));
	return gl_Vertex;
}

__device__ bool Shader::fragment(Vec3f bar, TGAColor &color, Vec3f normal, TGAColor diffuse) {
	Vec3f bn = (varying_nrm*bar).normalize();
	Vec2f uv = varying_uv * bar;

	mat<3, 3, float> A;
	A[0] = ndc_tri.col(1) - ndc_tri.col(0);
	A[1] = ndc_tri.col(2) - ndc_tri.col(0);
	A[2] = bn;

	mat<3, 3, float> AI = A.invert();

	Vec3f i = AI * Vec3f(varying_uv[0][1] - varying_uv[0][0], varying_uv[0][2] - varying_uv[0][0], 0);
	Vec3f j = AI * Vec3f(varying_uv[1][1] - varying_uv[1][0], varying_uv[1][2] - varying_uv[1][0], 0);

	mat<3, 3, float> B;
	B.set_col(0, i.normalize());
	B.set_col(1, j.normalize());
	B.set_col(2, bn);

	Vec3f n = (B*normal).normalize();//model->normal(uv)

	float diff = fmax(0.f, n*light_dir);
	//color = diffuse * diff;//model->diffuse(uv)*diff;

	return false;
}

Shader::Shader(Vec3f *light_dir)
{
	// make cuda copy
	this->light_dir = *light_dir;
}