#include "ModelBuffer.cuh"

ModelBuffer::ModelBuffer(Model *model)
{
	int dnverts = model->verts_.size();
	int dnfaces = model->faces_.size();
	int dnfacesElem = model->faces_[0].size();

	hipMalloc((void**)&nverts, sizeof(int));
	hipMalloc((void**)&nfaces, sizeof(int));
	hipMalloc((void**)&nfacesElem, sizeof(int));

	hipMemcpy(nverts, &dnverts, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(nfaces, &dnfaces, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(nfacesElem, &dnfacesElem, sizeof(int), hipMemcpyHostToDevice);
	
	unsigned long verts_bytes = model->verts_.size() * sizeof(Vec3f);
	unsigned long norms_bytes = model->norms_.size() * sizeof(Vec3f);
	unsigned long uv_bytes = model->uv_.size() * sizeof(Vec2f);

	hipMalloc((void**)&verts_, verts_bytes);
	hipMalloc((void**)&norms_, norms_bytes);
	hipMalloc((void**)&uv_, uv_bytes);

	hipMemcpy(verts_, model->verts_.data(), verts_bytes, hipMemcpyHostToDevice);
	hipMemcpy(norms_, model->norms_.data(), norms_bytes, hipMemcpyHostToDevice);
	hipMemcpy(uv_, model->uv_.data(), uv_bytes, hipMemcpyHostToDevice);

	unsigned long faces_bytes = model->faces_.size()*sizeof(Vec3i*);
	unsigned long faces_elements_bytes = model->faces_[0].size();

	int height = model->faces_.size();
	int width = dnfacesElem;

	// falatten 2d array from CPU to 1d array on GPU
	int size = height * width;
	Vec3i *flatten = (Vec3i*)malloc(size*sizeof(Vec3i));//new Vec3i[size];
	for (int h = 0; h < height; h++) {
		for (int w = 0; w < width; w++)
			flatten[width * h + w] = model->faces_[h][w];
	}

	hipMalloc((void**)&faces_, size*sizeof(Vec3i));
	hipMemcpy(faces_, flatten, size * sizeof(Vec3i), hipMemcpyHostToDevice);
	free(flatten);

	//init textures
	hipMalloc((void**)&diffuse_width, sizeof(int));
	hipMalloc((void**)&diffuse_height, sizeof(int));
	hipMalloc((void**)&diffuse_bytespp, sizeof(int));

	hipMemcpy(diffuse_width, &(model->diffusemap_.width), sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(diffuse_height, &(model->diffusemap_.height), sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(diffuse_bytespp, &(model->diffusemap_.bytespp), sizeof(int), hipMemcpyHostToDevice);

	unsigned long nbytes = model->diffusemap_.width * model->diffusemap_.height*model->diffusemap_.bytespp;
	hipMalloc((void**)&diffuse_data, nbytes);

	hipMemcpy(diffuse_data, model->diffusemap_.data, nbytes, hipMemcpyHostToDevice);
}

__device__ int* ModelBuffer::getNVerts()
{
	return this->nverts;
}
__device__ int* ModelBuffer::getNFaces()
{
	return this->nfaces;
}
__device__ int* ModelBuffer::getNFacesElem()
{
	return this->nfacesElem;
}

ModelBuffer::~ModelBuffer()
{
	//hipFree(verts_);
	//hipFree(norms_);
	//hipFree(uv_);
	//hipFree(faces_);
	//hipFree(nverts);
	//hipFree(nfaces);
	//hipFree(nfacesElem);
	//
	////free textute
	//hipFree(diffuse_data);
	//hipFree(diffuse_width);
	//hipFree(diffuse_height);
	//hipFree(diffuse_bytespp);
}

__device__ Color ModelBuffer::diffuse(Vec2i uv)
{
	//int x = uvf.x;
	//int y = uvf.y;
	//printf("x=%d, y=%d\n", uv.x, uv.y);
	if (uv.x < 0 || uv.y < 0 || uv.x >= *diffuse_width || uv.y >= *diffuse_height) {
	//	printf("NOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOO");
		return Color();
	}
	Color c(diffuse_data + (uv.x + uv.y * (*diffuse_width))*(*diffuse_bytespp), *diffuse_bytespp);
	c.alpha = 255;
	//printf("r=%d, g=%d, b=%d, a=%d\n", c.red, c.green, c.blue, c.alpha);
	return c;
}

__device__ Vec3f ModelBuffer::normal(int iface, int nthvert)
{
	int inx0 = (faces_[*nfacesElem * iface + nthvert])[2];
	return norms_[inx0].normalize();
}

__device__ Vec3f ModelBuffer::vert(int i)
{
	return verts_[i];
}

__device__ Vec3f ModelBuffer::vert(int iface, int nthvert)
{
	int idx = (faces_[*nfacesElem * iface + nthvert])[0];
	return verts_[idx];
}

__device__ Vec2i ModelBuffer::uv(int iface, int nthvert)
{
	//return uv_[faces_[iface][nthvert][1]];
	//int idx = (faces_[*nfacesElem * iface + nthvert])[0];
	//return uv_[idx];

	//int idx = faces_[iface][nthvert][1];
	int idx = (faces_[*nfacesElem * iface + nthvert])[1];
	//printf("x=%f, y=%f  w=d%d, h=%d\n", uv_[idx].x*(*diffuse_width), uv_[idx].y*(*diffuse_height), (*diffuse_width), (*diffuse_height));
	return Vec2i(uv_[idx].x*(*diffuse_width), uv_[idx].y*(*diffuse_height));
}

__device__ int ModelBuffer::face(int i, int idx) {
	return (faces_[*nfacesElem*i+idx])[0];
}

//__fmaf_rd:
//x * y + z

//width * i + j

//2d:
//data[y*w + x]
// x,y, width
// 3d:
//Flat[x + WIDTH * (y + DEPTH * z)] = Original[x, y, z]