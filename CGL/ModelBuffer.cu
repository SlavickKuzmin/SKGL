#include "ModelBuffer.cuh"

ModelBuffer::ModelBuffer(Model *model)
{
	int dnverts = model->verts_.size();
	int dnfaces = model->faces_.size();
	int dnfacesElem = model->faces_[0].size();

	hipMalloc((void**)&nverts, sizeof(int));
	hipMalloc((void**)&nfaces, sizeof(int));
	hipMalloc((void**)&nfacesElem, sizeof(int));

	hipMemcpy(nverts, &dnverts, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(nfaces, &dnfaces, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(nfacesElem, &dnfacesElem, sizeof(int), hipMemcpyHostToDevice);
	
	unsigned long verts_bytes = model->verts_.size() * sizeof(Vec3f);
	unsigned long norms_bytes = model->norms_.size() * sizeof(Vec3f);
	unsigned long uv_bytes = model->uv_.size() * sizeof(Vec2f);

	hipMalloc((void**)&verts_, verts_bytes);
	hipMalloc((void**)&norms_, norms_bytes);
	hipMalloc((void**)&uv_, uv_bytes);

	hipMemcpy(verts_, model->verts_.data(), verts_bytes, hipMemcpyHostToDevice);
	hipMemcpy(norms_, model->norms_.data(), norms_bytes, hipMemcpyHostToDevice);
	hipMemcpy(uv_, model->uv_.data(), uv_bytes, hipMemcpyHostToDevice);

	unsigned long faces_bytes = model->faces_.size()*sizeof(Vec3i*);
	unsigned long faces_elements_bytes = model->faces_[0].size();

	int height = model->faces_.size();
	int width = dnfacesElem;

	// falatten 2d array from CPU to 1d array on GPU
	int size = height * width;
	Vec3i *flatten = (Vec3i*)malloc(size*sizeof(Vec3i));//new Vec3i[size];
	for (int h = 0; h < height; h++) {
		for (int w = 0; w < width; w++)
			flatten[width * h + w] = model->faces_[h][w];
	}

	hipMalloc((void**)&faces_, size*sizeof(Vec3i));
	hipMemcpy(faces_, flatten, size * sizeof(Vec3i), hipMemcpyHostToDevice);
	free(flatten);

	//init diffuse texture
	// TODO possible leak
	TextureBuffer *diffText = new TextureBuffer(model->diffusemap_);
	hipMalloc((void**)&(this->diffuse_texture), sizeof(TextureBuffer));
	hipMemcpy(this->diffuse_texture, diffText, sizeof(TextureBuffer), hipMemcpyHostToDevice);

	// init nomal map texture
	TextureBuffer *normTex = new TextureBuffer(model->normalmap_);
	hipMalloc((void**)&(this->normal_map_texture), sizeof(TextureBuffer));
	hipMemcpy(this->normal_map_texture, normTex, sizeof(TextureBuffer), hipMemcpyHostToDevice);

	printf("Constr call\n");
}

__device__ int* ModelBuffer::getNVerts()
{
	return this->nverts;
}
__device__ int* ModelBuffer::getNFaces()
{
	return this->nfaces;
}
__device__ int* ModelBuffer::getNFacesElem()
{
	return this->nfacesElem;
}

ModelBuffer::~ModelBuffer()
{
	hipFree(verts_);
	hipFree(norms_);
	hipFree(uv_);
	hipFree(faces_);
	hipFree(nverts);
	hipFree(nfaces);
	hipFree(nfacesElem);
	
	// free texture
	hipFree(this->diffuse_texture);
	hipFree(this->normal_map_texture);
	printf("Destr call\n");
}

__device__ Color ModelBuffer::diffuse(Vec2f uvf)
{
	//if (uv.x < 0 || uv.y < 0 || uv.x >= diffuse_texture->getWidth() || uv.y >= diffuse_texture->getHeight()) {
	//	return Color();
	//}
	//Color c(diffuse_texture->texture_binary_data + (uv.x + uv.y*diffuse_texture->getWidth()*(diffuse_texture->getBytesApp()),
	//	diffuse_texture->getBytesApp());
	//c.alpha = 255;
	//return c;
	Vec2i uv(uvf[0] * diffuse_texture->getWidth(), uvf[1] * diffuse_texture->getHeight());
	return diffuse_texture->get(uv[0], uv[1]);
}

__device__ Vec3f ModelBuffer::normal(int iface, int nthvert)
{
	int inx0 = (faces_[*nfacesElem * iface + nthvert])[2];
	return norms_[inx0].normalize();
}
__device__ Vec3f ModelBuffer::normal(Vec2f uvf)
{
	Vec2i uv(uvf[0] * this->normal_map_texture->getWidth(), uvf[1] * this->normal_map_texture->getHeight());
	Color c = this->normal_map_texture->get(uv[0], uv[1]);
	Vec3f res;
	for (int i = 0; i < 3; i++)
		res[2 - i] = (float)c[i] / 255.f*2.f - 1.f;
	return res;
}

__device__ Vec3f ModelBuffer::vert(int i)
{
	return verts_[i];
}

__device__ Vec3f ModelBuffer::vert(int iface, int nthvert)
{
	int idx = (faces_[*nfacesElem * iface + nthvert])[0];
	return verts_[idx];
}

//__device__ Vec2i ModelBuffer::uv(int iface, int nthvert)
//{
//	int idx = (faces_[*nfacesElem * iface + nthvert])[1];
//	return Vec2i(uv_[idx].x*(diffuse_texture->getWidth()), uv_[idx].y*(diffuse_texture->getHeight()));
//}

__device__ Vec2f ModelBuffer::uv(int iface, int nthvert)
{
	//int idx = (faces_[*nfacesElem * iface + nthvert])[1];
	//return Vec2i(uv_[idx].x*(diffuse_texture->getWidth()), uv_[idx].y*(diffuse_texture->getHeight()));
	int idx = (faces_[*nfacesElem * iface + nthvert])[1];
	return uv_[idx];
}


__device__ int ModelBuffer::face(int i, int idx) {
	return (faces_[*nfacesElem*i+idx])[0];
}

//__fmaf_rd:
//x * y + z

//width * i + j

//2d:
//data[y*w + x]
// x,y, width
// 3d:
//Flat[x + WIDTH * (y + DEPTH * z)] = Original[x, y, z]