#include "ModelBuffer.cuh"

ModelBuffer::ModelBuffer(Model *model)
{
	int dnverts = model->verts_.size();
	int dnfaces = model->faces_.size();
	int dnfacesElem = model->faces_[0].size();

	hipMalloc((void**)&nverts, sizeof(int));
	hipMalloc((void**)&nfaces, sizeof(int));
	hipMalloc((void**)&nfacesElem, sizeof(int));

	hipMemcpy(nverts, &dnverts, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(nfaces, &dnfaces, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(nfacesElem, &dnfacesElem, sizeof(int), hipMemcpyHostToDevice);
	
	unsigned long verts_bytes = model->verts_.size() * sizeof(Vec3f);
	unsigned long norms_bytes = model->norms_.size() * sizeof(Vec3f);
	unsigned long uv_bytes = model->uv_.size() * sizeof(Vec2f);

	hipMalloc((void**)&verts_, verts_bytes);
	hipMalloc((void**)&norms_, norms_bytes);
	hipMalloc((void**)&uv_, uv_bytes);

	hipMemcpy(verts_, model->verts_.data(), verts_bytes, hipMemcpyHostToDevice);
	hipMemcpy(norms_, model->norms_.data(), norms_bytes, hipMemcpyHostToDevice);
	hipMemcpy(uv_, model->uv_.data(), uv_bytes, hipMemcpyHostToDevice);

	unsigned long faces_bytes = model->faces_.size()*sizeof(Vec3i*);
	unsigned long faces_elements_bytes = model->faces_[0].size();

	int height = model->faces_.size();
	int width = dnfacesElem;

	// falatten 2d array from CPU to 1d array on GPU
	int size = height * width;
	Vec3i *flatten = (Vec3i*)malloc(size*sizeof(Vec3i));//new Vec3i[size];
	for (int h = 0; h < height; h++) {
		for (int w = 0; w < width; w++)
			flatten[width * h + w] = model->faces_[h][w];
	}

	hipMalloc((void**)&faces_, size*sizeof(Vec3i));
	hipMemcpy(faces_, flatten, size * sizeof(Vec3i), hipMemcpyHostToDevice);
	free(flatten);
}

__device__ int* ModelBuffer::getNVerts()
{
	return this->nverts;
}
__device__ int* ModelBuffer::getNFaces()
{
	return this->nfaces;
}
__device__ int* ModelBuffer::getNFacesElem()
{
	return this->nfacesElem;
}

ModelBuffer::~ModelBuffer()
{
	hipFree(verts_);
	hipFree(norms_);
	hipFree(uv_);
	hipFree(faces_);
	hipFree(nverts);
	hipFree(nfaces);
	hipFree(nfacesElem);
}

__device__ Vec3f ModelBuffer::normal(int iface, int nthvert)
{
	int inx0 = (faces_[*nfacesElem * iface + nthvert])[2];
	//int idx = faces_[iface][nthvert][2];
	return norms_[inx0].normalize();
}

__device__ Vec3f ModelBuffer::vert(int i)
{
	return verts_[i];
}

__device__ Vec3f ModelBuffer::vert(int iface, int nthvert)
{
	int idx = (faces_[*nfacesElem * iface + nthvert])[0];
	return verts_[idx];
}

__device__ Vec2f ModelBuffer::uv(int iface, int nthvert)
{
	//return uv_[faces_[iface][nthvert][1]];
	int idx = (faces_[*nfacesElem * iface + nthvert])[0];
	return uv_[idx];
}

__device__ int ModelBuffer::face(int i, int idx) {
	return (faces_[*nfacesElem*i+idx])[0];
}

//width * i + j

//2d:
//data[y*w + x]
// x,y, width
// 3d:
//Flat[x + WIDTH * (y + DEPTH * z)] = Original[x, y, z]