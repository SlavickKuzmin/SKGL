#include "hip/hip_runtime.h"
#include "RenderOnGPU.cuh"

gl::RenderOnGPU::RenderOnGPU(Model *model, Screen *screen)
{
	this->screen = screen;

	this->width = screen->GetWidth();
	this->height = screen->GetHeight();

	ModelBuffer *mb = new ModelBuffer(model);
	// make model
	// TODO possible leak
	hipMalloc((void**)&(this->model), sizeof(ModelBuffer));
	hipMemcpy(this->model, mb, sizeof(ModelBuffer), hipMemcpyHostToDevice);
	//this->model = mb;
	this->m = model;

	zbuffer = new float[width*height];
	for (int i = width * height; i--; zbuffer[i] = -std::numeric_limits<float>::max());

	//int *zBufferGPU;
	hipMalloc((void**)&zBufferGPU, width*height * sizeof(float));
	
	threads_size = 5022; // diablo_pose
	//threads_size = 2492; // african_head
	int* arr = splitByThreads(m->nfaces(), threads_size);

	//int *cArr;
	hipMalloc((void**)&cArr, sizeof(int)*(threads_size + 1));
	hipMemcpy(cArr, arr, sizeof(int)*(threads_size + 1), hipMemcpyHostToDevice);
	free(arr);

	// save pinch
	this->pinch = screen->pixels->pitch;

	// save host pixels
	this->h_pixels = screen->pixels->pixels;

	//alloc device pixels
	int size = height * pinch;
	hipMalloc((void**)&this->d_pixels, size);
	hipMemcpy(this->d_pixels, this->h_pixels, size, hipMemcpyHostToDevice);
}

gl::RenderOnGPU::~RenderOnGPU()
{
	delete model;
	//{ // dump z-buffer (debugging purposes only)
	//	TGAImage zbimage(width, height, TGAImage::GRAYSCALE);
	//	for (int i = 0; i < width; i++) {
	//		for (int j = 0; j < height; j++) {
	//			zbimage.set(i, j, TGAColor(zbuffer[i + j * width], 1, 1));
	//		}
	//	}
	//	zbimage.flip_vertically(); // i want to have the origin at the left bottom corner of the image
	//	zbimage.write_tga_file("D:\\zbuffer.tga");
	//}
	delete[] zbuffer;
	hipFree(zBufferGPU);
	hipFree(cArr);
	hipFree(d_pixels);
}

//==============================================================================================================
struct Shader : public gl::IShader {
	mat<2, 3, float> *varying_uv;  // triangle uv coordinates, written by the vertex shader, read by the fragment shader
	mat<4, 3, float> *varying_tri; // triangle coordinates (clip coordinates), written by VS, read by FS
	mat<3, 3, float> *varying_nrm; // normal per vertex to be interpolated by FS
	mat<3, 3, float> *ndc_tri;     // triangle in normalized device coordinates

	gl::ModelBuffer *model;
	Matrix Projection;
	Matrix ModelView;
	Vec3f light_dir;

	__device__ Shader(gl::ModelBuffer *mb, Matrix &Projection, Matrix &ModelView, Vec3f &light_dir)
	{
		this->model = mb;
		this->Projection = Projection;
		this->ModelView = ModelView;
		this->light_dir = light_dir;

		//hipMalloc((void**)&varying_uv, sizeof(mat<2, 3, float>));
		//hipMalloc((void**)&varying_tri, sizeof(mat<4, 3, float>));
		//hipMalloc((void**)&varying_nrm, sizeof(mat<3, 3, float>));
		//hipMalloc((void**)&ndc_tri, sizeof(mat<3, 3, float>));
		mat<2, 3, float> varying_uv;  // triangle uv coordinates, written by the vertex shader, read by the fragment shader
		mat<4, 3, float> varying_tri; // triangle coordinates (clip coordinates), written by VS, read by FS
		mat<3, 3, float> varying_nrm; // normal per vertex to be interpolated by FS
		mat<3, 3, float> ndc_tri;     // triangle in normalized device coordinates
		this->varying_uv = &varying_uv;
		this->varying_tri = &varying_tri;
		this->varying_nrm = &varying_nrm;
		this->ndc_tri = &ndc_tri;
	}

	__device__ virtual Vec4f vertex(int iface, int nthvert) {
		Vec2f uvRes = model->uv(iface, nthvert);
		varying_uv->set_col(nthvert, &uvRes);
		vec<3, float> proj1Res = proj<3>(((Projection*ModelView).invert_transpose()*embed<4>(model->normal(iface, nthvert), 0.f)));
		varying_nrm->set_col(nthvert, &proj1Res);
		Vec4f gl_Vertex = Projection * ModelView*embed<4>(model->vert(iface, nthvert));
		varying_tri->set_col(nthvert, &gl_Vertex);
		vec<3, float> proj2Res = proj<3>(gl_Vertex / gl_Vertex[3]);
		ndc_tri->set_col(nthvert, &proj2Res);
		return gl_Vertex;
	}

	__device__ virtual bool fragment(Vec3f bar, gl::Color::Device &color) {
		Vec3f bn = (*varying_nrm*bar).normalize();
		Vec2f uv = *varying_uv * bar;

		mat<3, 3, float> A;
		A[0] = ndc_tri->col(1) - ndc_tri->col(0);
		A[1] = ndc_tri->col(2) - ndc_tri->col(0);
		A[2] = bn;

		mat<3, 3, float> AI = A.invert();

		Vec3f i = AI * Vec3f((*varying_uv)[0][1] - (*varying_uv)[0][0], (*varying_uv)[0][2] - (*varying_uv)[0][0], 0);
		Vec3f j = AI * Vec3f((*varying_uv)[1][1] - (*varying_uv)[1][0], (*varying_uv)[1][2] - (*varying_uv)[1][0], 0);

		mat<3, 3, float> B;
		B.set_col(0, &i.normalize());
		B.set_col(1, &j.normalize());
		B.set_col(2, &bn);

		Vec3f n = (B*model->normal(uv)).normalize();

		float diff = fmaxf(0.f, n*light_dir);
		color = model->diffuse(uv)*diff;

		return false;
	}
};
//==============================================================================================================

__device__ void part(void* pixels, int pinch, int width, int height, gl::ModelBuffer *mb,
	int first, int last, float *zbuffer, float ra, float command, gl::RenderMode mode)
{
	Vec3f light_dir(1, 1, 1);
	Vec3f       eye(command, ra, 1);
	Vec3f    center(0, 0, 0);
	Vec3f        up(0, 1, 0);

	Matrix ModelView;
	Matrix Viewport;
	Matrix Projection;

	gl::camera::lookat(ModelView, eye, center, up);
	gl::camera::viewport(Viewport, width / 8, height / 8, width * 3 / 4, height * 3 / 4);
	gl::camera::projection(Projection, -1.f / (eye - center).norm());
	light_dir = proj<3>((Projection*ModelView*embed<4>(light_dir, 0.f))).normalize();

	if (mode == gl::RenderMode::Shaders)
	{
		//ModelBuffer *mb, Matrix &Projection, Matrix &ModelView, Vec3f &light_dir
		//triangle_s(mat<4, 3, float> &clipc, IShader &shader, void* pixels, int pinch, float *zbuffer, Matrix &Viewport)
		Shader shader(mb, Projection, ModelView, light_dir);
		for (int i = first; i < last; i++) {
			for (int j = 0; j < 3; j++) {
				shader.vertex(i, j);
			}
			////	//triangle(shader.varying_tri, shader, frame, zbuffer);
			gl::draw::triangle_s(shader.varying_tri, &shader, pixels, pinch, zbuffer, Viewport, ra);
		}
	}
	else if (mode == gl::RenderMode::Filled)
	{
		for (int i = first; i < last; i++) {
			Vec2i screen_coords[3];
			for (int j = 0; j < 3; j++) {
				Vec3f v = mb->vert(mb->face(i, j));
				Matrix result = Viewport * Projection*ModelView*Matrix(v);
				screen_coords[j] = Vec2f(result[0][0] / result[3][0], result[1][0] / result[3][0]);
			}
			gl::draw::triangle(screen_coords[0], screen_coords[1], screen_coords[2], pixels, pinch, &gl::Color::Device(255, 0, 0));
		}
	}
	else if (mode == gl::RenderMode::Wire)
	{
		for (int i = first; i < last; i++) {
			Vec3i screen_coords[3];
			for (int j = 0; j < 3; j++) {
				Vec3f v = mb->vert(mb->face(i, j));
				Matrix result = Viewport*Projection*ModelView*Matrix(v);
				screen_coords[j] = Vec3f(result[0][0]/result[3][0], result[1][0]/result[3][0], result[2][0]/result[3][0]);
			}
			gl::draw::line(screen_coords[0].x, screen_coords[0].y, screen_coords[1].x, screen_coords[1].y, pixels, pinch, &gl::Color::Device(255, 0, 0));
			gl::draw::line(screen_coords[1].x, screen_coords[1].y, screen_coords[2].x, screen_coords[2].y, pixels, pinch, &gl::Color::Device(0, 255, 0));
			gl::draw::line(screen_coords[2].x, screen_coords[2].y, screen_coords[0].x, screen_coords[0].y, pixels, pinch, &gl::Color::Device(0, 0, 255));
		}
	}
	else if (mode == gl::RenderMode::ShadersWithWire)
	{
		//ModelBuffer *mb, Matrix &Projection, Matrix &ModelView, Vec3f &light_dir
		//triangle_s(mat<4, 3, float> &clipc, IShader &shader, void* pixels, int pinch, float *zbuffer, Matrix &Viewport)
		Shader shader(mb, Projection, ModelView, light_dir);
		for (int i = first; i < last; i++) {
			Vec3i screen_coords[3];
			for (int j = 0; j < 3; j++) {
				shader.vertex(i, j);
				Vec3f v = mb->vert(mb->face(i, j));
				Matrix result = Viewport * Projection*ModelView*Matrix(v);
				screen_coords[j] = Vec3f(result[0][0] / result[3][0], result[1][0] / result[3][0], result[2][0] / result[3][0]);
			}
			////	//triangle(shader.varying_tri, shader, frame, zbuffer);
			gl::draw::triangle_s(shader.varying_tri, &shader, pixels, pinch, zbuffer, Viewport, ra);

			gl::draw::line(screen_coords[0].x, screen_coords[0].y, screen_coords[1].x, screen_coords[1].y, pixels, pinch, &gl::Color::Device(255, 0, 0));
			gl::draw::line(screen_coords[1].x, screen_coords[1].y, screen_coords[2].x, screen_coords[2].y, pixels, pinch, &gl::Color::Device(0, 255, 0));
			gl::draw::line(screen_coords[2].x, screen_coords[2].y, screen_coords[0].x, screen_coords[0].y, pixels, pinch, &gl::Color::Device(0, 0, 255));
		}
	}
}

int* gl::splitByThreads(int model, int parts)
{
	int array_size = parts + 1;
	int* part_array = (int*)malloc(array_size*sizeof(int));
	int partInOneThread = model / parts;
	int lastElementSize = (model - (partInOneThread*parts)) + partInOneThread;

	int counter = -partInOneThread;
	for (int i = 0; i < array_size - 1; i++)
	{
		counter = counter + partInOneThread;
		part_array[i] = counter;
	}
	part_array[array_size - 1] = counter + lastElementSize;

	return part_array;
}

__device__ void debugPrint(int *arr, int size)
{
	for (int i = 0; i < size - 1; i++)
	{
		printf("[%d] s=%d, e=%d ", i, arr[i], arr[i + 1]);
	}
	printf("\n");
}

__global__ void SplitByMPs(void* pixels, int pinch, int width, int height, gl::ModelBuffer *mb, int threads_size,
	int *arr, float *zbuffer, float ra, float command, gl::RenderMode mode)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	//printf("size=%d\n", threads_size);
	
	if (idx < threads_size + 1)
	{
		//debugPrint(arr, threads_size + 1);
		//printf("idx=%d\n", idx);
		part(pixels, pinch, width, height, mb, arr[idx], arr[idx + 1], zbuffer, ra, command, mode);
	}

}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
	}
}

void gl::RenderOnGPU::refresh(float direction, float command, RenderMode mode)
{
	clock_t begin = clock();

	hipMemcpy(this->d_pixels, this->h_pixels, height * pinch, hipMemcpyHostToDevice);
	hipMemcpy(zBufferGPU, zbuffer, width*height * sizeof(float), hipMemcpyHostToDevice);

	SplitByMPs <<<128, 128 >>> (this->d_pixels, pinch, width, height, model, threads_size, cArr, zBufferGPU, direction, command, mode);

	//printf("model=%d, threads_size=%d\n",m->nfaces(), threads_size);
	//RenderSadersMode <<<128, 128 >>> (this->d_pixels, pinch, width, height, model, threads_size, cArr, zBufferGPU, direction, command);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	hipMemcpy(this->h_pixels, this->d_pixels, height * pinch, hipMemcpyDeviceToHost);

	clock_t end = clock();
	this->renderFrameTime = float(end - begin) / CLOCKS_PER_SEC;
	printf("time: %lf\n", this->renderFrameTime);
}

float& gl::RenderOnGPU::GetRenderFrameTime()
{
	return this->renderFrameTime;
}