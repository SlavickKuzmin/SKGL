#include "hip/hip_runtime.h"
#include "RenderOnGPU.cuh"

RenderOnGPU::RenderOnGPU(Model *model, int width, int height)
{
	this->width = width;
	this->height = height;

	ModelBuffer *mb = new ModelBuffer(model);
	// make model
	this->model = mb;
	this->m = model;

	zbuffer = new int[width*height];
	for (int i = 0; i < width*height; i++) {
		zbuffer[i] = std::numeric_limits<int>::min();
	}
}

RenderOnGPU::~RenderOnGPU()
{
	delete model;
	delete[] zbuffer;
}

__device__ void part(void* pixels, int pinch, int width, int height, ModelBuffer &mb, int first, int last, int *zbuffer)
{
	//printf("f=%d, l=%d\n", first, last);
	
	// old
	Vec3f light_dir(0, 0, -1);//todo remove it
	const int depth = 255;//todo it too
	for (int i = first; i < last; i++) {
		Vec2i screen_coords[3];
		Vec3f world_coords[3];
		for (int j = 0; j < 3; j++) {
			Vec3f v = mb.vert(mb.face(i, j));
			screen_coords[j] = Vec2i((v.x + 1.)*width / 2., (v.y + 1.)*height / 2.);
			world_coords[j] = v;
		}
		Vec3f n = (world_coords[2] - world_coords[0]) ^ (world_coords[1] - world_coords[0]);
		n.normalize();
		float intensity = n * light_dir;
		if (intensity > 0) {
			Color col;
			col.alpha = 255;
			col.red = 255 * intensity;
			col.green = 0;
			col.blue = 0;
			triangle(screen_coords[0], screen_coords[1], screen_coords[2], pixels, pinch, &col);
		}
	}
	// old without light
	//////printf("r=%d, g=%d, b=%d\n", col.red, col.green, col.blue);
	//for (int i = first; i < last; i++) {
	//	Vec2i screen_coords[3];
	//	for (int j = 0; j < 3; j++) {
	//		Vec3f world_coords = mb.vert(mb.face(i,j));
	//		screen_coords[j] = Vec2i((world_coords.x + 1.)*width / 2., (world_coords.y + 1.)*height / 2.);
	//	}
	//	
	//	triangle(screen_coords[0], screen_coords[1], screen_coords[2], pixels, pinch, &col);
	//	// Linew render
	//	/*line(screen_coords[0].x, screen_coords[0].y, screen_coords[1].x, screen_coords[1].y, pixels, pinch, &col);
	//	line(screen_coords[1].x, screen_coords[1].y, screen_coords[2].x, screen_coords[2].y, pixels, pinch, &col);
	//	line(screen_coords[2].x, screen_coords[2].y, screen_coords[0].x, screen_coords[0].y, pixels, pinch, &col);*/
	//}
	////printf("r=%d, g=%d, b=%d\n", col.red, col.green, col.blue);

	// new
	//Color col;
	//col.alpha = 255;
	//col.red = 255;
	//col.green = 0;
	//col.blue = 0;
	//Vec3f light_dir(0, 0, -1);//todo remove it
	//const int depth = 255;//todo it too
	//for (int i = first; i < last; i++) {
	//	Vec3i screen_coords[3];
	//	Vec3f world_coords[3];
	//	for (int j = 0; j < 3; j++) {
	//		Vec3f v = mb.vert(mb.face(i, j));
	//		screen_coords[j] = Vec3i((v.x + 1.)*width / 2., (v.y + 1.)*height / 2., (v.z + 1.)*depth / 2.);
	//		world_coords[j] = v;
	//	}
	//	Vec3f n = (world_coords[2] - world_coords[0])^(world_coords[1] - world_coords[0]);
	//	n.normalize();
	//	float intensity = n * light_dir;
	//	if (intensity > 0) {
	//		triangleZBuf(screen_coords[0], screen_coords[1], screen_coords[2], pixels, pinch, &col, zbuffer);
	//	}
	//}
}

int* splitByThreads(int model, int parts)
{
	int array_size = parts + 1;
	int* part_array = (int*)malloc(array_size*sizeof(int));
	int partInOneThread = model / parts;
	int lastElementSize = (model - (partInOneThread*parts)) + partInOneThread;

	int counter = -partInOneThread;
	for (int i = 0; i < array_size - 1; i++)
	{
		counter = counter + partInOneThread;
		part_array[i] = counter;
	}
	part_array[array_size - 1] = counter + lastElementSize;

	return part_array;
}

__device__ void debugPrint(int *arr, int size)
{
	for (int i = 0; i < size - 1; i++)
	{
		printf("[%d] s=%d, e=%d ", i, arr[i], arr[i + 1]);
	}
	printf("\n");
}

__global__ void draw(void* pixels, int pinch, int width, int height, ModelBuffer mb, int threads_size, int *arr, int *zbuffer)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	//printf("size=%d\n", threads_size);
	
	if (idx < threads_size + 1)
	{
		//debugPrint(arr, threads_size + 1);
		//printf("idx=%d\n", idx);
		part(pixels, pinch, width, height, mb, arr[idx], arr[idx + 1], zbuffer);
	}

}


#define M 2

void RenderOnGPU::refresh(void* pixels, int pinch, int width, int height)
{
	void *gpuPixels;

	int size = height * pinch;
	hipMalloc((void**)&gpuPixels, size);
	hipMemcpy(gpuPixels, pixels, size, hipMemcpyHostToDevice);

	int *zBufferGPU;
	hipMalloc((void**)&zBufferGPU, width*height * sizeof(int));
	hipMemcpy(zBufferGPU, zbuffer, width*height * sizeof(int), hipMemcpyHostToDevice);

	clock_t begin = clock();

	//// parts is 7, res array size 8
	//int* arr = splitByThreads(5022, 20);
	//debugPrint(arr, 21);
	printf(".");
	int threads_size = 60;
	int* arr = splitByThreads(m->nfaces(), threads_size);

	int *cArr;
	hipMalloc((void**)&cArr, sizeof(int)*(threads_size+1));
	hipMemcpy(cArr, arr, sizeof(int)*(threads_size + 1), hipMemcpyHostToDevice);

	draw <<<64, 2 >>> (gpuPixels, pinch, width, height, *model, threads_size, cArr, zBufferGPU);
	hipDeviceSynchronize();

	free(arr);
	hipFree(cArr);
	hipFree(zBufferGPU);
	
	clock_t end = clock();
	double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;

	//printf("time: %lf\n", elapsed_secs);

	hipMemcpy(pixels, gpuPixels, size, hipMemcpyDeviceToHost);
	hipFree(gpuPixels);
}