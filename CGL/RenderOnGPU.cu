#include "hip/hip_runtime.h"
#include "RenderOnGPU.cuh"


RenderOnGPU::RenderOnGPU(ModelBuffer *model, int width, int height)
{
	this->width = width;
	this->height = height;

	// make model
	this->model = model;
}

RenderOnGPU::~RenderOnGPU()
{
	delete model;
}

__device__ void part(void* pixels, int pinch, int width, int height, ModelBuffer &mb, int first, int last)
{
	for (int i = first; i < last; i++) {
		for (int j = 0; j < 3; j++) {
			Vec3f v0 = mb.vert(mb.face(i, j));
			Vec3f v1 = mb.vert(mb.face(i, (j + 1) % 3));
			int x0 = (v0.x + 1.f)*width / 2.f;
			int y0 = (v0.y + 1.f)*height / 2.f;
			int x1 = (v1.x + 1.f)*width / 2.f;
			int y1 = (v1.y + 1.f)*height / 2.f;
			line(x0, y0, x1, y1, pixels, pinch);
		}
	}
}

__global__ void draw(void* pixels, int pinch, int width, int height, ModelBuffer mb)
{
	//printf("s");
	int idx = blockIdx.x;
	//printf("idx=%d\n", idx);
	//5022
	// 0 => 3000
	// 3000 => 5022
	//printf("size=%d\n", *(mb.nfaces));
	printf(".");
	//for (int i = 0; i < *(mb.nfaces); i++) {
	//	for (int j = 0; j < 3; j++) {
	//		Vec3f v0 = mb.vert(mb.face(i, j));
	//		Vec3f v1 = mb.vert(mb.face(i, (j + 1) % 3));
	//		int x0 = (v0.x + 1.f)*width / 2.f;
	//		int y0 = (v0.y + 1.f)*height / 2.f;
	//		int x1 = (v1.x + 1.f)*width / 2.f;
	//		int y1 = (v1.y + 1.f)*height / 2.f;
	//		line(x0, y0, x1, y1, pixels, pinch);
	//	}
	//}
	if(idx == 0)
	{
		part(pixels,pinch,width,height,mb, 0, 1000);
	}
	else if (idx == 1)
	{
		part(pixels, pinch, width, height, mb, 1000, 2000);
	}
	else if (idx == 2)
	{
		part(pixels, pinch, width, height, mb, 2000, 3000);
	}
	else if (idx == 3)
	{
		part(pixels, pinch, width, height, mb, 3000, 4000);
	}
	else if (idx == 4)
	{
		part(pixels, pinch, width, height, mb, 4000, 5022);
	}
	//printf("e");
}

void RenderOnGPU::refresh(void* pixels, int pinch, int width, int height)
{
	void *gpuPixels;

	int size = height * pinch;
	hipMalloc((void**)&gpuPixels, size);
	hipMemcpy(gpuPixels, pixels, size, hipMemcpyHostToDevice);

	//drawModel<<<1,1>>>(this->shader, mb, ModelView, Projection, Viewport, zBufferGPU, pixels, pinch, width, height);
	
	/*drawModel << <1, 1 >> > (this->shader, mBuf, ModelView, Projection, Viewport, zBufferGPU, pixels, pinch, width, height);
	hipDeviceSynchronize();*/
	draw<<<5, 1>>> (gpuPixels, pinch, width, height, *model);
	hipDeviceSynchronize();

	//for (int i = 0; i < model->nfaces(); i++) {
	//	std::vector<int> face = model->face(i);
	//	for (int j = 0; j < 3; j++) {
	//		Vec3f v0 = model->vert(face[j]);
	//		Vec3f v1 = model->vert(face[(j + 1) % 3]);
	//		int x0 = (v0.x + 1.)*width / 2.;
	//		int y0 = (v0.y + 1.)*height / 2.;
	//		int x1 = (v1.x + 1.)*width / 2.;
	//		int y1 = (v1.y + 1.)*height / 2.;
	//		line << <1, 1 >> > (x0, y0, x1, y1, gpuPixels, pinch);
	//		hipDeviceSynchronize();
	//	}
	//}

	//printf(".");

	hipMemcpy(pixels, gpuPixels, size, hipMemcpyDeviceToHost);
	hipFree(gpuPixels);
	//printf("start-");

	//for (int i = 0; i < model->nfaces(); i++) {
	//	for (int j = 0; j < 3; j++) {
	//		//this->shader->vertex(i, j, ModelView, Projection);
	//	}
	//	//triangle(this->shader->varying_tri, *shader, this->renderer, zbuffer, Viewport);
	//}

	//printf("end ");
}