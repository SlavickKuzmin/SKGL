#include "hip/hip_runtime.h"
#include "cuda_gl.cuh"

const int width = 800;
const int height = 800;

__device__ void viewport(int x, int y, int w, int h, Matrix &Viewport) {
	Viewport = Matrix::identity();
	Viewport[0][3] = x + w / 2.f;
	Viewport[1][3] = y + h / 2.f;
	Viewport[2][3] = 1.f;
	Viewport[0][0] = w / 2.f;
	Viewport[1][1] = h / 2.f;
	Viewport[2][2] = 0;
}

__device__ void projection(float coeff, Matrix &Projection) {
	Projection = Matrix::identity();
	Projection[3][2] = coeff;
}

__device__ void lookat(Vec3f eye, Vec3f center, Vec3f up, Matrix &ModelView) {
	Vec3f z = (eye - center).normalize();
	Vec3f x = cross(up, z).normalize();
	Vec3f y = cross(z, x).normalize();
	Matrix Minv = Matrix::identity();
	Matrix Tr = Matrix::identity();
	for (int i = 0; i < 3; i++) {
		Minv[0][i] = x[i];
		Minv[1][i] = y[i];
		Minv[2][i] = z[i];
		Tr[i][3] = -center[i];
	}
	ModelView = Minv * Tr;
}

__device__ Vec3f barycentric(Vec2f A, Vec2f B, Vec2f C, Vec2i P) {
	Vec3f s[2];
	for (int i = 2; i--; ) {
		s[i][0] = C[i] - A[i];
		s[i][1] = B[i] - A[i];
		s[i][2] = A[i] - P[i];
	}
	Vec3f u = cross(s[0], s[1]);
	if (abs(u[2]) > 1e-2) // dont forget that u[2] is integer. If it is zero then triangle ABC is degenerate
		return Vec3f(1.f - (u.x + u.y) / u.z, u.y / u.z, u.x / u.z);
	return Vec3f(-1, 1, 1); // in this case generate negative coordinates, it will be thrown away by the rasterizator
}

__device__ void triangle(mat<4, 3, float> &clipc, Shader &shader, void *pixels, int pinch, float *zbuffer, Matrix &Viewport) {
	mat<3, 4, float> pts = (Viewport*clipc).transpose(); // transposed to ease access to each of the points
    mat<3, 2, float> pts2;
	for (int i = 0; i < 3; i++) pts2[i] = proj<2>(pts[i] / pts[i][3]);

	Vec2f bboxmin(FLT_MAX, FLT_MAX);
	Vec2f bboxmax(-FLT_MAX, -FLT_MAX);
	Vec2f clamp(width - 1, height - 1); // with, height
	for (int i = 0; i < 3; i++) {
		for (int j = 0; j < 2; j++) {
			bboxmin[j] = fmax(0.f, fmin(bboxmin[j], pts2[i][j]));
			bboxmax[j] = fmin(clamp[j], fmax(bboxmax[j], pts2[i][j]));
		}
	}
	Vec2i P;
	Color color;
	for (P.x = bboxmin.x; P.x <= bboxmax.x; P.x++) {
		for (P.y = bboxmin.y; P.y <= bboxmax.y; P.y++) {
			Vec3f bc_screen = barycentric(pts2[0], pts2[1], pts2[2], P);
			Vec3f bc_clip = Vec3f(bc_screen.x / pts[0][3], bc_screen.y / pts[1][3], bc_screen.z / pts[2][3]);
			bc_clip = bc_clip / (bc_clip.x + bc_clip.y + bc_clip.z);
			float frag_depth = clipc[2] * bc_clip;
			if (bc_screen.x < 0 || bc_screen.y < 0 || bc_screen.z<0 || zbuffer[P.x + P.y*width]>frag_depth) continue; // 800 - image with
			bool discard = true;//= shader.fragment(bc_clip, TGAColor());
			if (!discard) {
				zbuffer[P.x + P.y*width] = frag_depth;//image.width
				//setPixel(pixels, pinch, P.x, P.y, color);
			}
		}
	}
}

