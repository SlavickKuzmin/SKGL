#include "TextureBuffer.cuh"

TextureBuffer::TextureBuffer(TGAImage &image)
{
	hipMalloc((void**)&d_pWidth, sizeof(int));
	hipMalloc((void**)&d_pHeight, sizeof(int));
	hipMalloc((void**)&d_pBytesApp, sizeof(int));

	hipMemcpy(d_pWidth, &(image.width), sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_pHeight, &(image.height), sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_pBytesApp, &(image.bytespp), sizeof(int), hipMemcpyHostToDevice);

	unsigned long nbytes = image.width * image.height*image.bytespp;
	hipMalloc((void**)&texture_binary_data, nbytes);

	hipMemcpy(texture_binary_data, image.data, nbytes, hipMemcpyHostToDevice);
	printf("Texture constructor called\n");
}

TextureBuffer::~TextureBuffer()
{
	hipFree(texture_binary_data);
	hipFree(d_pWidth);
	hipFree(d_pHeight);
	hipFree(d_pBytesApp);
	printf("Texture destructor called\n");
}

__device__ int TextureBuffer::getWidth()
{
	return *(this->d_pWidth);
}
__device__ int TextureBuffer::getHeight()
{
	return *(this->d_pHeight);
}
__device__ int TextureBuffer::getBytesApp()
{
	return *(this->d_pBytesApp);
}