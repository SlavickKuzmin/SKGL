/*
 * Description: Contains a API implementation for loading texture image
 * in GPU memory and manupulation with it.
 * Created by Viacheslav Kuzmin 2019
 */

#include "TextureBuffer.cuh"

// Construct a class from image stored on CPU memory, 
// allocate GPU memory.
TextureBuffer::TextureBuffer(TGAImage &image)
{
	// Allocate device memory for data storage.
	hipMalloc((void**)&d_pWidth, sizeof(int));
	hipMalloc((void**)&d_pHeight, sizeof(int));
	hipMalloc((void**)&d_pBytesApp, sizeof(int));

	// Copy memory from host to device.
	hipMemcpy(d_pWidth, &(image.width), sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_pHeight, &(image.height), sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_pBytesApp, &(image.bytespp), sizeof(int), hipMemcpyHostToDevice);

	unsigned long nbytes = image.width * image.height*image.bytespp;
	hipMalloc((void**)&texture_binary_data, nbytes);

	hipMemcpy(texture_binary_data, image.data, nbytes, hipMemcpyHostToDevice);
}

// Free all used GPU memory.
TextureBuffer::~TextureBuffer()
{
	// Free all alocated device memory.
	hipFree(texture_binary_data);
	hipFree(d_pWidth);
	hipFree(d_pHeight);
	hipFree(d_pBytesApp);
}

// Gets a texture width.
__device__ int TextureBuffer::getWidth()
{
	return *(this->d_pWidth);
}

// Gets a texture height.
__device__ int TextureBuffer::getHeight()
{
	return *(this->d_pHeight);
}

// Gets a texture bytes app.
__device__ int TextureBuffer::getBytesApp()
{
	return *(this->d_pBytesApp);
}

// Gets a texture pixel color from specific position (x and y coords).
__device__ Color TextureBuffer::get(int x, int y)
{
	// Validate input parameters.
	if (!texture_binary_data || x < 0 || y < 0 || x >= getWidth() || y >= getHeight()) {
		return Color();
	}

	// return a requested color.
	return Color(texture_binary_data + (x + y * getWidth())*getBytesApp(), getBytesApp());
}