/*
 * Description: Contains a API implementation for loading texture image
 * in GPU memory and manupulation with it.
 * Created by Viacheslav Kuzmin 2019
 */

#include "TextureBuffer.cuh"

// Construct a class from image stored on CPU memory, 
// allocate GPU memory.
gl::TextureBuffer::TextureBuffer(TGAImage &image)
{
	// Allocate device memory for data storage.
	hipMalloc((void**)&d_pWidth, sizeof(int));
	hipMalloc((void**)&d_pHeight, sizeof(int));
	hipMalloc((void**)&d_pBytesApp, sizeof(int));

	// Copy memory from host to device.
	hipMemcpy(d_pWidth, &(image.width), sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_pHeight, &(image.height), sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_pBytesApp, &(image.bytespp), sizeof(int), hipMemcpyHostToDevice);

	unsigned long nbytes = image.width * image.height*image.bytespp;
	hipMalloc((void**)&texture_binary_data, nbytes);

	hipMemcpy(texture_binary_data, image.data, nbytes, hipMemcpyHostToDevice);
}

// Free all used GPU memory.
gl::TextureBuffer::~TextureBuffer()
{
	// Free all alocated device memory.
	hipFree(texture_binary_data);
	hipFree(d_pWidth);
	hipFree(d_pHeight);
	hipFree(d_pBytesApp);
}

// Gets a texture width.
__device__ int gl::TextureBuffer::getWidth()
{
	return *(this->d_pWidth);
}

// Gets a texture height.
__device__ int gl::TextureBuffer::getHeight()
{
	return *(this->d_pHeight);
}

// Gets a texture bytes app.
__device__ int gl::TextureBuffer::getBytesApp()
{
	return *(this->d_pBytesApp);
}

// Gets a texture pixel color from specific position (x and y coords).
__device__ gl::Color::Device gl::TextureBuffer::get(int x, int y)
{
	// Validate input parameters.
	if (!texture_binary_data || x < 0 || y < 0 || x >= getWidth() || y >= getHeight()) {
		return gl::Color::Device();
	}

	// return a requested color.
	return gl::Color::Device(texture_binary_data + (x + y * getWidth())*getBytesApp(), getBytesApp());
}